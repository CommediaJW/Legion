#include "feature_storage.cuh"
#include "feature_storage_impl.cuh"
#include <iostream>

#include <unordered_set>
#include <algorithm>
#include <random>
#include <assert.h>
#include <unistd.h>

class CompleteFeatureStorage : public FeatureStorage{
public: 
    CompleteFeatureStorage(){
    }

    virtual ~CompleteFeatureStorage(){};

    void Build(BuildInfo* info, int in_memory_mode) override {
        int32_t partition_count = info->partition_count;
        total_num_nodes_ = info->total_num_nodes;
        float_feature_len_ = info->float_feature_len;
        float* host_float_feature = info->host_float_feature;

        if(in_memory_mode){
            hipHostGetDevicePointer(&float_feature_, host_float_feature, 0);
        }
        cudaCheckError();

        training_set_num_.resize(partition_count);
        training_set_ids_.resize(partition_count);
        training_labels_.resize(partition_count);

        validation_set_num_.resize(partition_count);
        validation_set_ids_.resize(partition_count);
        validation_labels_.resize(partition_count);

        testing_set_num_.resize(partition_count);
        testing_set_ids_.resize(partition_count);
        testing_labels_.resize(partition_count);

        partition_count_ = partition_count;

        for(int32_t i = 0; i < partition_count_; i++){
            int32_t part_id = i;
            training_set_num_[part_id] = info->training_set_num[part_id];
            validation_set_num_[part_id] = info->validation_set_num[part_id];
            testing_set_num_[part_id] = info->testing_set_num[part_id];

            hipSetDevice(part_id);
            cudaCheckError();

            int32_t* train_ids;
            hipMalloc(&train_ids, training_set_num_[part_id] * sizeof(int32_t));
            hipMemcpy(train_ids, info->training_set_ids[part_id].data(), training_set_num_[part_id] * sizeof(int32_t), hipMemcpyHostToDevice);
            training_set_ids_[part_id] = train_ids;
            cudaCheckError();

            int32_t* valid_ids;
            hipMalloc(&valid_ids, validation_set_num_[part_id] * sizeof(int32_t));
            hipMemcpy(valid_ids, info->validation_set_ids[part_id].data(), validation_set_num_[part_id] * sizeof(int32_t), hipMemcpyHostToDevice);
            validation_set_ids_[part_id] = valid_ids;
            cudaCheckError();

            int32_t* test_ids;
            hipMalloc(&test_ids, testing_set_num_[part_id] * sizeof(int32_t));
            hipMemcpy(test_ids, info->testing_set_ids[part_id].data(), testing_set_num_[part_id] * sizeof(int32_t), hipMemcpyHostToDevice);
            testing_set_ids_[part_id] = test_ids;
            cudaCheckError();

            int32_t* train_labels;
            hipMalloc(&train_labels, training_set_num_[part_id] * sizeof(int32_t));
            hipMemcpy(train_labels, info->training_labels[part_id].data(), training_set_num_[part_id] * sizeof(int32_t), hipMemcpyHostToDevice);
            training_labels_[part_id] = train_labels;
            cudaCheckError();

            int32_t* valid_labels;
            hipMalloc(&valid_labels, validation_set_num_[part_id] * sizeof(int32_t));
            hipMemcpy(valid_labels, info->validation_labels[part_id].data(), validation_set_num_[part_id] * sizeof(int32_t), hipMemcpyHostToDevice);
            validation_labels_[part_id] = valid_labels;
            cudaCheckError();

            int32_t* test_labels;
            hipMalloc(&test_labels, testing_set_num_[part_id] * sizeof(int32_t));
            hipMemcpy(test_labels, info->testing_labels[part_id].data(), testing_set_num_[part_id] * sizeof(int32_t), hipMemcpyHostToDevice);
            testing_labels_[part_id] = test_labels;
            cudaCheckError();

        }

    };

    void Finalize() override {
        hipHostFree(float_feature_);
        for(int32_t i = 0; i < partition_count_; i++){
            hipSetDevice(i);
            hipFree(training_set_ids_[i]);
            hipFree(validation_set_ids_[i]);
            hipFree(testing_set_ids_[i]);
            hipFree(training_labels_[i]);
            hipFree(validation_labels_[i]);
            hipFree(testing_labels_[i]);
        }
    }

    int32_t* GetTrainingSetIds(int32_t part_id) const override {
        return training_set_ids_[part_id];
    }
    int32_t* GetValidationSetIds(int32_t part_id) const override {
        return validation_set_ids_[part_id];
    }
    int32_t* GetTestingSetIds(int32_t part_id) const override {
        return testing_set_ids_[part_id];
    }

	int32_t* GetTrainingLabels(int32_t part_id) const override {
        return training_labels_[part_id];
    };
    int32_t* GetValidationLabels(int32_t part_id) const override {
        return validation_labels_[part_id];
    }
    int32_t* GetTestingLabels(int32_t part_id) const override {
        return testing_labels_[part_id];
    }

    int32_t TrainingSetSize(int32_t part_id) const override {
        return training_set_num_[part_id];
    }
    int32_t ValidationSetSize(int32_t part_id) const override {
        return validation_set_num_[part_id];
    }
    int32_t TestingSetSize(int32_t part_id) const override {
        return testing_set_num_[part_id];
    }

    int32_t TotalNodeNum() const override {
        return total_num_nodes_;
    }

    float* GetAllFloatFeature() const override {
        return float_feature_;
    }
    int32_t GetFloatFeatureLen() const override {
        return float_feature_len_;
    }

    void IOSubmit(int32_t* sampled_ids, int32_t* cache_index,
                  int32_t* node_counter, float* dst_float_buffer,
                  int32_t op_id, int32_t dev_id, hipStream_t strm_hdl) override {
		//TODO
    }

    void IOComplete() override {
        //TODO
    }

private:
    std::vector<int> training_set_num_;
    std::vector<int> validation_set_num_;
    std::vector<int> testing_set_num_;

    std::vector<int32_t*> training_set_ids_;
    std::vector<int32_t*> validation_set_ids_;
    std::vector<int32_t*> testing_set_ids_;

    std::vector<int32_t*> training_labels_;
    std::vector<int32_t*> validation_labels_;
    std::vector<int32_t*> testing_labels_;

    int32_t partition_count_;
    int32_t total_num_nodes_;
    float* float_feature_;
    int32_t float_feature_len_;

    friend FeatureStorage* NewCompleteFeatureStorage();
};

extern "C" 
FeatureStorage* NewCompleteFeatureStorage(){
    CompleteFeatureStorage* ret = new CompleteFeatureStorage();
    return ret;
}
